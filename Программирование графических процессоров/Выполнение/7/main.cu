#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/extrema.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

//Заполнение матрицы псевдослучайным числами
void fill_rand(float *arr, int rows, int cols)
{
	//Создание генератора псевдослучайных чисел
	hiprandGenerator_t rng;
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);

	//Инициализация
	hiprandSetPseudoRandomGeneratorSeed(rng, (unsigned long long) clock());

	//Заполнение матрицы
	hiprandGenerateUniform(rng, arr, rows * cols);
}

//Перемножение матриц
void mmul(const float *arrA, const float *arrB, float *result, const int m, const int k, const int n)
{
	int lda = m, ldb = k, ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	//Создание дескриптора для сuBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//Непосредственно умножение
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, arrA, lda, arrB, ldb, beta, result, ldc);

	//Удаление дескриптора
	hipblasDestroy(handle);
}

void print_matrix(const float *arr, int rows, int cols)
{
	for(int i = 0; i < rows; ++i) {
		for(int j = 0; j < cols; ++j) {
			printf(" %f ", arr[j * rows + i]);
		}
		printf("\n");
	}
	printf("\n");
}

int main(int argc, char *argv[])
{
	int rows, cols;
	rows = atoi(argv[1]);
	cols = atoi(argv[1]);

	printf("Matrix's size = %d\n", atoi(argv[1]));
	thrust::device_vector<float> d_A(rows * cols), d_B(rows * cols), d_C(rows * cols);

	//Заполнение матрицы А
	fill_rand(thrust::raw_pointer_cast(&d_A[0]), rows, cols);
	//Заполнение матрицы В
	fill_rand(thrust::raw_pointer_cast(&d_B[0]), rows, cols);

	thrust::host_vector<float> h_A = d_A;
	thrust::host_vector<float> h_B = d_B;
	//Вывод маассивов перед умножением
	printf("A = ");
	print_matrix(thrust::raw_pointer_cast(&h_A[0]), rows, cols);
	printf("B = ");
	print_matrix(thrust::raw_pointer_cast(&h_B[0]), rows, cols);

	//Перемножение матриц
	mmul(thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]), thrust::raw_pointer_cast(&d_C[0]), rows, cols, cols);

	thrust::host_vector<float> h_C = d_C;

	//Вывод результатов
	printf("C = ");
	print_matrix(thrust::raw_pointer_cast(&h_C[0]), rows, cols);

	//Поиск максимального значения
	thrust::device_vector<float>::iterator iter = thrust::max_element(d_C.begin(), d_C.end());
	float max_val = *iter;
	printf("Max element = %f\n", max_val);

	//Поиск минимального значения
	thrust::device_vector<float>::iterator iter2 = thrust::min_element(d_C.begin(), d_C.end());
	float min_val = *iter2;
	printf("Min element = %f\n", min_val);

	return 0;
}