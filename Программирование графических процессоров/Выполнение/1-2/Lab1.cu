#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void gTest(float *a, float *b)
{
	a[threadIdx.x + blockDim.x * blockIdx.x] = (float)(threadIdx.x + blockDim.x * blockIdx.x);
	b[threadIdx.x + blockDim.x * blockIdx.x] = 2.0;
	a[threadIdx.x + blockDim.x * blockIdx.x] += b[threadIdx.x + blockDim.x * blockIdx.x];
}

int main(int argc, char *argv[])
{
	float *vec, *vec2, *result;
	int size = 8388608;
	int threads_per_block = atoi(argv[1]);
	int num_of_blocks = (int)(size / threads_per_block);

	result = (float*) malloc(size * sizeof(float));
	hipMalloc((void**)&vec, size * sizeof(float));
	hipMalloc((void**)&vec2, size * sizeof(float));
	gTest <<< dim3(num_of_blocks), dim3(threads_per_block) >>> (vec, vec2);

	hipDeviceSynchronize();
	hipMemcpy(result, vec, size * sizeof(float), hipMemcpyDeviceToHost);

	free(result);
	hipFree(vec);
	hipFree(vec2);

	return 0;
}