#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#define BLOCK_SIZE 32

//����� �������
void show(float* a, int n)
{
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            fprintf(stdout, "%g\t", a[j + i * n]);
        }
        fprintf(stdout, "\n");
    }
    fprintf(stdout, "\n\n\n");
}

__global__ void matMult_global(float *a, float *b, int n, float* c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; //���� ����� -> ���� ������� �������
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    float sum = 0.0f;
    for (int k = 0; k < n; ++k)
        sum += a[i * n + k] * b[k * n + j];
    c[i * n + j] = sum;

}

__global__ void matMult_shared(float *a, float *b, int n, float* c)
{
    int aBegin = n * BLOCK_SIZE * blockIdx.y; //������ ������ ������ ���������� �, ������� ������������ ����
    int aEnd = aBegin + n - 1; //������ ����� ���������� �, ������� ������������ ����
    int aStep = BLOCK_SIZE; //��� ��� �������� ������ �

    int bBegin = BLOCK_SIZE * blockIdx.x; //������ ������ ������ ���������� �, ������� ������������ ����
    int bStep = BLOCK_SIZE * n; //��� ��� �������� �������� B

    float sum = 0.0f; //���������� �������� ����������

    for (int sub_A = aBegin, sub_B = bBegin; sub_A <= aEnd; sub_A += aStep, sub_B += bStep) {
        //��������� ���������� ������ ��� ���������
        __shared__ float buffer_A[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float buffer_B[BLOCK_SIZE][BLOCK_SIZE];

        //�������� ��������� � � � �� ���������� ������ � �����������
		buffer_A[threadIdx.y][threadIdx.x] = a[sub_A + n * threadIdx.y + threadIdx.x];
        buffer_B[threadIdx.y][threadIdx.x] = b[sub_B + n * threadIdx.y + threadIdx.x];

        __syncthreads(); // ��������, ��� ���������� ��������� ���������

        //������������ ���� ������
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += buffer_A[threadIdx.x][k] * buffer_B[k][threadIdx.x];
        __syncthreads(); // ��������, ��� ���������� ������ ������ �� �����
    }

    int cBegin = n * BLOCK_SIZE * blockIdx.x + BLOCK_SIZE * blockIdx.x; //������ ��������������� �������� � ���������� ������
    c[cBegin + n * threadIdx.y + threadIdx.x] = sum; //������ �������� � ���������� ������
}

int main (int argc, char *  argv [])
{
    //���� ����������� �������
    if(argc < 2){
        fprintf(stderr, "������� ������ �������: ");
        return -1;
    }

    //�������� ������� ������
    int N = atoi(argv[1]);
	if(N % BLOCK_SIZE) {
        fprintf(stderr, "�������� ������\n");
        return -1;
    }

    int numBytes = N * N * sizeof(float);

    //������������� ��������� �������
    //��������� ������ �� host
    float *a = (float*)malloc(numBytes); //������� �
    float *b = (float*)malloc(numBytes); //������� B
    float *c = (float*)malloc(numBytes); //������� �

    //���������� ������
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            int k = N * i + j;
            a[k] = 2.0f;
            b[k] = 1.0f;
        }

    float *adevice, *bdevice, *cdevice;

    //��������� ������ �� GPU
    hipMalloc((void**)&adevice, numBytes);
	hipMalloc((void**)&bdevice, numBytes);
    hipMalloc((void**)&cdevice, numBytes);


    //������� ����� ����� � ������
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    //����������� ������ � � B c host �� device
    //������������� ���������� �� CPU -> �������� �� GPU
    hipMemcpy(adevice, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdevice, b, numBytes, hipMemcpyHostToDevice);

    //������ �������
    matMult_global<<<blocks, threads>>>(adevice, bdevice, N, cdevice);
    hipDeviceSynchronize();

    //�����������, ����������� ������� � � device �� host
    //���������� ������� -> ���������
    hipMemcpy(c, cdevice, numBytes, hipMemcpyDeviceToHost);

	hipMemcpy (adevice, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy (bdevice, b, numBytes, hipMemcpyHostToDevice);

    matMult_shared<<<blocks, threads>>>(adevice, bdevice, N, cdevice);
    hipDeviceSynchronize();
    hipMemcpy(c, cdevice, numBytes, hipMemcpyDeviceToHost);

    //������������ ������ �� GPU � CPU
    hipFree(adevice);
    hipFree(bdevice);
    hipFree(cdevice);

    free(a);
    free(b);
    free(c);

    return 0;
}