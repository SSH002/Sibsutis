#include <hip/hip_runtime.h>
#include <cstdio>
#include <sys/time.h>
#include <iostream>
#include <cmath>

#define ROWS 2500
#define COLS 2500
#define BLOCK_SIZE 32

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if(_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error %s at line %d in file %s\n",\
		hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	} }

double wtime()
{
	struct timeval t;
	gettimeofday(&t,NULL);
	return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

void print_matrix(float *a, size_t rows, size_t cols)
{
	for (int i = 0; i < rows * cols; i++) {
		if(i % cols == 0 ) {
			printf("\n");
		}
		printf(" %g", a[i]);
	}
	printf("\n");
}

__global__ void gDgemm(float* a, float *b, float *c, int count_rows, int count_cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row < count_rows && col < count_cols) {
		float s = 0.0;
		for (int k = 0; k < COLS; k++) {
			s += a[row * COLS + k] * b[k * ROWS + col];
		}
		c[row * count_cols + col] = s;
	}
}
void init_matrix(float **a, float **b, float **c)
{
	CUDA_CHECK_RETURN(hipHostAlloc(a, ROWS * COLS * sizeof(float), hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipHostAlloc(b, COLS * ROWS * sizeof(float), hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipHostAlloc(c, ROWS * ROWS * sizeof(float), hipHostMallocDefault));

	srand(time(NULL));

	for (int i = 0; i < ROWS * COLS ; i++) {
		(*a)[i] = rand() % 10+1;
		(*b)[i] = rand() % 10+1;
	}
}

void dgemm_with_streams(float *host_a, float *host_b, float *host_c, int streams) {
	float *device_a, *device_b, *device_c;
	int si, count_rows = ceil(ROWS / (float)streams), size_chunk = count_rows * COLS;

	hipStream_t stream[streams];
	for (int i = 0; i < streams; i++) {
		hipStreamCreate(&stream[i]);
	}

	CUDA_CHECK_RETURN(hipMalloc(&device_a, ROWS * COLS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc(&device_b, COLS * ROWS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc(&device_c, ROWS * ROWS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy(device_b, host_b, COLS * ROWS * sizeof(float), hipMemcpyHostToDevice));

	dim3 gridDim = dim3(ceil(ROWS / (float)BLOCK_SIZE), ceil(count_rows / (float)BLOCK_SIZE), 1);
	dim3 blockDim = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);

	double time = wtime();
	for (si = 0; si < streams; ++si) {
		float * dev_a_matrix_part = device_a + si * size_chunk;
		float * dev_c_matrix_part = device_c + si * count_rows * ROWS;
		float * host_a_matrix_part = host_a + si * size_chunk;
		float * host_c_matrix_part = host_c + si * count_rows * ROWS;

		if( count_rows * streams > ROWS ) {
			int diff = (count_rows * streams - ROWS);
			CUDA_CHECK_RETURN(hipMemcpyAsync(dev_a_matrix_part, host_a_matrix_part, (count_rows - diff) * COLS * sizeof(float), hipMemcpyHostToDevice, stream[si]));
			gDgemm <<< gridDim, blockDim, 0, stream[si]>>> (device_a + si * size_chunk, device_b, device_c + si * count_rows * ROWS, count_rows - diff, ROWS);
			CUDA_CHECK_RETURN(hipMemcpyAsync(host_c_matrix_part, dev_c_matrix_part, (count_rows - diff) * ROWS * sizeof(float), hipMemcpyDeviceToHost, stream[si]));
		} else {
			CUDA_CHECK_RETURN(hipMemcpyAsync(dev_a_matrix_part, host_a_matrix_part, size_chunk * sizeof(float), hipMemcpyHostToDevice, stream[si]));
			gDgemm <<< gridDim, blockDim,  0, stream[si]>>> (device_a + si * size_chunk, device_b, device_c + si * count_rows * ROWS, count_rows, ROWS);
			CUDA_CHECK_RETURN(hipMemcpyAsync(host_c_matrix_part, dev_c_matrix_part, count_rows * ROWS * sizeof(float), hipMemcpyDeviceToHost, stream[si]));
		}
	}

	hipDeviceSynchronize();

	time = wtime() - time;
	printf("Time: %f ms\n", time);

	//print_matrix(host_c, ROWS, COLS);

	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
}

int main ()
{
	int streams = 1;
	float *host_a = NULL;
	float *host_b = NULL;
	float *host_c = NULL;

	init_matrix(&host_a, &host_b, &host_c);

	printf("Work with %d thread :\n", streams);
	dgemm_with_streams(host_a, host_b, host_c, streams);

	printf("Enter the number of threads to start: ");
	scanf("%d", &streams);
	printf("Work with %d thread :\n", streams);

	dgemm_with_streams(host_a, host_b, host_c, streams);
	printf("\n");

	return 0;
}