#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE (1024 * 1024)

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat = value;\
	if (_m_cudaStat != hipSuccess) {\
		fprintf(stderr, "Error %s at line %d in file %s\n",\
		hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
		exit(1);\
	} }

float cuda_pageable(int size, bool flag) {
	int *host_a, *dev_a;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	host_a = (int*)malloc(size * sizeof(int));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a, size * sizeof(float)));

	hipEventRecord(start, 0);

	if(flag)
		CUDA_CHECK_RETURN(hipMemcpy(dev_a, host_a, size * sizeof(float), hipMemcpyHostToDevice));

	if(!flag)
		CUDA_CHECK_RETURN(hipMemcpy(host_a, dev_a, size * sizeof(float), hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	free(host_a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}

float cuda_page_locked(int size, bool flag) {
	int *host_a, *dev_a;
	float elapsedTime;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	CUDA_CHECK_RETURN(hipHostAlloc((void**)&host_a, size * sizeof(float), hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_a, size * sizeof(float)));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	if(flag)
		CUDA_CHECK_RETURN(hipMemcpy(dev_a, host_a, size * sizeof(float), hipMemcpyHostToDevice));

	if(!flag)
		CUDA_CHECK_RETURN(hipMemcpy(host_a, dev_a, size * sizeof(float), hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipHostFree(host_a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsedTime;
}

int main(int argc, char *argv[])
{
	float elapsedTime;
	int numbMB = atoi(argv[1]);

	int size = SIZE * numbMB;

	elapsedTime = cuda_pageable(size, true);
	printf("\nPageable memory:\n\n[CPU -> GPU] = %f ms\n", elapsedTime);

	elapsedTime = cuda_pageable(size, false);
	printf("[GPU -> CPU] =  %f ms\n", elapsedTime);

	elapsedTime = cuda_page_locked(size, true);
	printf("\nPage-locked memory:\n\n[CPU -> GPU] = %f ms\n", elapsedTime);

	elapsedTime = cuda_page_locked(size, false);
	printf("[GPU -> CPU] = %f ms\n\n", elapsedTime);

	return 0;
}